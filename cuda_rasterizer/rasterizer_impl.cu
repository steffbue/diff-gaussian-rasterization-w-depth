#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "rasterizer_impl.h"
#include <iostream>
#include <fstream>
#include <algorithm>
#include <numeric>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

#include "auxiliary.h"
#include "forward.h"
#include "backward.h"

void checkValidCUDAPointer(void *ptr) 
{
	hipPointerAttribute_t attr;
	hipError_t err = hipPointerGetAttributes(&attr, ptr);
	if (err != hipSuccess) {
		printf("Invalid or unrecognized CUDA pointer: %s\n", hipGetErrorString(err));
	} else {
		// attr.type tells you if it's device, host, or managed memory
		printf("Pointer type: %d\n", attr.type);
	}
}

#define CUDA_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}

// Helper function to find the next-highest bit of the MSB
// on the CPU.
uint32_t getHigherMsb(uint32_t n)
{
	uint32_t msb = sizeof(n) * 4;
	uint32_t step = msb;
	while (step > 1)
	{
		step /= 2;
		if (n >> msb)
			msb += step;
		else
			msb -= step;
	}
	if (n >> msb)
		msb++;
	return msb;
}

// Wrapper method to call auxiliary coarse frustum containment test.
// Mark all Gaussians that pass it.
__global__ void checkFrustum(int P,
	const float* orig_points,
	const float* viewmatrix,
	const float* projmatrix,
	bool* present)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	float3 p_view;
	present[idx] = in_frustum(idx, orig_points, viewmatrix, projmatrix, false, p_view);
}

// Generates one key/value pair for all Gaussian / tile overlaps. 
// Run once per Gaussian (1:N mapping).
__global__ void duplicateWithKeys(
	int P,
	const float2* points_xy,
	const float* depths,
	const uint32_t* offsets,
	uint64_t* gaussian_keys_unsorted,
	uint32_t* gaussian_values_unsorted,
	int* radii,
	dim3 grid)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Generate no key/value pair for invisible Gaussians
	if (radii[idx] > 0)
	{
		// Find this Gaussian's offset in buffer for writing keys/values.
		uint32_t off = (idx == 0) ? 0 : offsets[idx - 1];
		uint2 rect_min, rect_max;

		getRect(points_xy[idx], radii[idx], rect_min, rect_max, grid);

		// For each tile that the bounding rect overlaps, emit a 
		// key/value pair. The key is |  tile ID  |      depth      |,
		// and the value is the ID of the Gaussian. Sorting the values 
		// with this key yields Gaussian IDs in a list, such that they
		// are first sorted by tile and then by depth. 
		for (int y = rect_min.y; y < rect_max.y; y++)
		{
			for (int x = rect_min.x; x < rect_max.x; x++)
			{
				uint64_t key = y * grid.x + x;
				key <<= 32;
				key |= *((uint32_t*)&depths[idx]);
				gaussian_keys_unsorted[off] = key;
				gaussian_values_unsorted[off] = idx;
				off++;
			}
		}
	}
}

// Check keys to see if it is at the start/end of one tile's range in 
// the full sorted list. If yes, write start/end of this tile. 
// Run once per instanced (duplicated) Gaussian ID.
__global__ void identifyTileRanges(int L, uint64_t* point_list_keys, uint2* ranges)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= L)
		return;

	// Read tile ID from key. Update start/end of tile range if at limit.
	uint64_t key = point_list_keys[idx];
	uint32_t currtile = key >> 32;
	if (idx == 0)
		ranges[currtile].x = 0;
	else
	{
		uint32_t prevtile = point_list_keys[idx - 1] >> 32;
		if (currtile != prevtile)
		{
			ranges[prevtile].y = idx;
			ranges[currtile].x = idx;
		}
	}
	if (idx == L - 1)
		ranges[currtile].y = L;
}

// Mark Gaussians as visible/invisible, based on view frustum testing
void CudaRasterizer::Rasterizer::markVisible(
	int P,
	float* means3D,
	float* viewmatrix,
	float* projmatrix,
	bool* present)
{
	checkFrustum << <(P + 255) / 256, 256 >> > (
		P,
		means3D,
		viewmatrix, projmatrix,
		present);
}

CudaRasterizer::GeometryState CudaRasterizer::GeometryState::fromChunk(char*& chunk, size_t P)
{
	GeometryState geom;
	obtain(chunk, geom.depths, P, 128);
	obtain(chunk, geom.clamped, P * 3, 128);
	obtain(chunk, geom.internal_radii, P, 128);
	obtain(chunk, geom.means2D, P, 128);
	obtain(chunk, geom.cov3D, P * 6, 128);
	obtain(chunk, geom.conic_opacity, P, 128);
	obtain(chunk, geom.rgb, P * 3, 128);
	obtain(chunk, geom.tiles_touched, P, 128);
	hipcub::DeviceScan::InclusiveSum(nullptr, geom.scan_size, geom.tiles_touched, geom.tiles_touched, P);
	obtain(chunk, geom.scanning_space, geom.scan_size, 128);
	obtain(chunk, geom.point_offsets, P, 128);
	obtain(chunk, geom.bbx_min, P, 128);
	obtain(chunk, geom.bbx_max, P, 128);

	return geom;
}

CudaRasterizer::FlowState CudaRasterizer::FlowState::fromChunk(char*& chunk, size_t P)
{
	FlowState flow;
	obtain(chunk, flow.prev_means2D, P, 128);
	obtain(chunk, flow.prev_cov2D_opacity, P, 128);
	obtain(chunk, flow.sqrt_conic, P, 128);
	obtain(chunk, flow.prev_sqrt_cov2D, P, 128);
	return flow;
}

CudaRasterizer::ImageState CudaRasterizer::ImageState::fromChunk(char*& chunk, size_t N)
{
	ImageState img;
	obtain(chunk, img.accum_alpha, N, 128);
	obtain(chunk, img.n_contrib, N, 128);
	obtain(chunk, img.ranges, N, 128);
	return img;
}

CudaRasterizer::BinningState CudaRasterizer::BinningState::fromChunk(char*& chunk, size_t P)
{
	BinningState binning;
	obtain(chunk, binning.point_list, P, 128);
	obtain(chunk, binning.point_list_unsorted, P, 128);
	obtain(chunk, binning.point_list_keys, P, 128);
	obtain(chunk, binning.point_list_keys_unsorted, P, 128);
	hipcub::DeviceRadixSort::SortPairs(
		nullptr, binning.sorting_size,
		binning.point_list_keys_unsorted, binning.point_list_keys,
		binning.point_list_unsorted, binning.point_list, P);
	obtain(chunk, binning.list_sorting_space, binning.sorting_size, 128);
	return binning;
}

CudaRasterizer::GaussianHeaderState CudaRasterizer::GaussianHeaderState::fromChunk(char*& chunk, size_t P)
{
	GaussianHeaderState header;
	obtain(chunk, header.bbx_min, P, 128);
	obtain(chunk, header.bbx_max, P, 128);
	obtain(chunk, header.cache_offset, P, 128);
	return header;
}

CudaRasterizer::CacheState CudaRasterizer::CacheState::fromChunk(char*& chunk, size_t N)
{
	CacheState cache;
	obtain(chunk, cache.t_value, N, 128);
	obtain(chunk, cache.g_value, N, 128);
	return cache;
}

// Forward rendering procedure for differentiable rasterization
// of Gaussians.
int CudaRasterizer::Rasterizer::forward(
	std::function<char* (size_t)> geometryBuffer,
	std::function<char* (size_t)> binningBuffer,
	std::function<char* (size_t)> imageBuffer,
	const int P, int D, int M,
	const float* background,
	const int width, int height,
	const float* means3D,
	const float* shs,
	const float* colors_precomp,
	const float* opacities,
	const float* scales,
	const float scale_modifier,
	const float* rotations,
	const float* cov3D_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const float* cam_pos,
	const float tan_fovx, float tan_fovy,
	const bool prefiltered,
	float* out_color,
	float* out_depth,
	int* radii)
{
	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	size_t chunk_size = required<GeometryState>(P);
	char* chunkptr = geometryBuffer(chunk_size);
	GeometryState geomState = GeometryState::fromChunk(chunkptr, P);

	if (radii == nullptr)
	{
		radii = geomState.internal_radii;
	}

	dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	dim3 block(BLOCK_X, BLOCK_Y, 1);

	// Dynamically resize image-based auxiliary buffers during training
	int img_chunk_size = required<ImageState>(width * height);
	char* img_chunkptr = imageBuffer(img_chunk_size);
	ImageState imgState = ImageState::fromChunk(img_chunkptr, width * height);

	if (NUM_CHANNELS != 3 && colors_precomp == nullptr)
	{
		throw std::runtime_error("For non-RGB, provide precomputed Gaussian colors!");
	}

	// Run preprocessing per-Gaussian (transformation, bounding, conversion of SHs to RGB)
	FORWARD::preprocess(
		P, D, M,
		means3D,
		(glm::vec3*)scales,
		scale_modifier,
		(glm::vec4*)rotations,
		opacities,
		shs,
		geomState.clamped,
		cov3D_precomp,
		colors_precomp,
		viewmatrix, projmatrix,
		(glm::vec3*)cam_pos,
		width, height,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
		radii,
		geomState.means2D,
		geomState.depths,
		geomState.cov3D,
		geomState.rgb,
		geomState.conic_opacity,
		geomState.bbx_min,
		geomState.bbx_max,
		tile_grid,
		geomState.tiles_touched,
		prefiltered
	);

	// Compute prefix sum over full list of touched tile counts by Gaussians
	// E.g., [2, 3, 0, 2, 1] -> [2, 5, 5, 7, 8]
	hipcub::DeviceScan::InclusiveSum(geomState.scanning_space, geomState.scan_size,
		geomState.tiles_touched, geomState.point_offsets, P);

	// Retrieve total number of Gaussian instances to launch and resize aux buffers
	int num_rendered;
	CUDA_CHECK(hipMemcpy(&num_rendered, geomState.point_offsets + P - 1, sizeof(int), hipMemcpyDeviceToHost));

	int binning_chunk_size = required<BinningState>(num_rendered);
	char* binning_chunkptr = binningBuffer(binning_chunk_size);
	BinningState binningState = BinningState::fromChunk(binning_chunkptr, num_rendered);

	// For each instance to be rendered, produce adequate [ tile | depth ] key 
	// and corresponding dublicated Gaussian indices to be sorted
	duplicateWithKeys << <(P + 255) / 256, 256 >> > (
		P,
		geomState.means2D,
		geomState.depths,
		geomState.point_offsets,
		binningState.point_list_keys_unsorted,
		binningState.point_list_unsorted,
		radii,
		tile_grid
		);

	int bit = getHigherMsb(tile_grid.x * tile_grid.y);

	// Sort complete list of (duplicated) Gaussian indices by keys
	hipcub::DeviceRadixSort::SortPairs(
		binningState.list_sorting_space,
		binningState.sorting_size,
		binningState.point_list_keys_unsorted, binningState.point_list_keys,
		binningState.point_list_unsorted, binningState.point_list,
		num_rendered, 0, 32 + bit);

	CUDA_CHECK(hipMemset(imgState.ranges, 0, tile_grid.x * tile_grid.y * sizeof(uint2)));

	// Identify start and end of per-tile workloads in sorted list
	if (num_rendered > 0)
		identifyTileRanges << <(num_rendered + 255) / 256, 256 >> > (
			num_rendered,
			binningState.point_list_keys,
			imgState.ranges
			);

	// Let each tile blend its range of Gaussians independently in parallel
	const float* feature_ptr = colors_precomp != nullptr ? colors_precomp : geomState.rgb;
	FORWARD::render(
		tile_grid, block,
		imgState.ranges,
		binningState.point_list,
		width, height,
		geomState.means2D,
		feature_ptr,
		geomState.conic_opacity,
		imgState.accum_alpha,
		imgState.n_contrib,
		background,
		out_color,
		geomState.depths,
		out_depth);

	return num_rendered;
}

// Produce necessary gradients for optimization, corresponding
// to forward render pass
void CudaRasterizer::Rasterizer::backward(
	const int P, int D, int M, int R,
	const float* background,
	const int width, int height,
	const float* means3D,
	const float* shs,
	const float* colors_precomp,
	const float* scales,
	const float scale_modifier,
	const float* rotations,
	const float* cov3D_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const float* campos,
	const float tan_fovx, float tan_fovy,
	const int* radii,
	char* geom_buffer,
	char* binning_buffer,
	char* img_buffer,
	const float* dL_dpix,
	float* dL_dmean2D,
	float* dL_dconic,
	float* dL_dopacity,
	float* dL_dcolor,
	float* dL_dmean3D,
	float* dL_dcov3D,
	float* dL_dsh,
	float* dL_dscale,
	float* dL_drot)
{
	GeometryState geomState = GeometryState::fromChunk(geom_buffer, P);
	BinningState binningState = BinningState::fromChunk(binning_buffer, R);
	ImageState imgState = ImageState::fromChunk(img_buffer, width * height);

	if (radii == nullptr)
	{
		radii = geomState.internal_radii;
	}

	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	const dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	const dim3 block(BLOCK_X, BLOCK_Y, 1);

	// Compute loss gradients w.r.t. 2D mean position, conic matrix,
	// opacity and RGB of Gaussians from per-pixel loss gradients.
	// If we were given precomputed colors and not SHs, use them.
	const float* color_ptr = (colors_precomp != nullptr) ? colors_precomp : geomState.rgb;
	BACKWARD::render(
		tile_grid,
		block,
		imgState.ranges,
		binningState.point_list,
		width, height,
		background,
		geomState.means2D,
		geomState.conic_opacity,
		color_ptr,
		imgState.accum_alpha,
		imgState.n_contrib,
		dL_dpix,
		(float3*)dL_dmean2D,
		(float4*)dL_dconic,
		dL_dopacity,
		dL_dcolor);

	// Take care of the rest of preprocessing. Was the precomputed covariance
	// given to us or a scales/rot pair? If precomputed, pass that. If not,
	// use the one we computed ourselves.
	const float* cov3D_ptr = (cov3D_precomp != nullptr) ? cov3D_precomp : geomState.cov3D;
	BACKWARD::preprocess(P, D, M,
		(float3*)means3D,
		radii,
		shs,
		geomState.clamped,
		(glm::vec3*)scales,
		(glm::vec4*)rotations,
		scale_modifier,
		cov3D_ptr,
		viewmatrix,
		projmatrix,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
		(glm::vec3*)campos,
		(float3*)dL_dmean2D,
		dL_dconic,
		(glm::vec3*)dL_dmean3D,
		dL_dcolor,
		dL_dcov3D,
		dL_dsh,
		(glm::vec3*)dL_dscale,
		(glm::vec4*)dL_drot);
}

// Create a cache of transmittance and Gaussian function values per pixel.
void CudaRasterizer::Rasterizer::createCache(
	std::function<char* (size_t)> geometryBuffer,
	std::function<char* (size_t)> binningBuffer,
	std::function<char* (size_t)> imageBuffer,
	std::function<char* (size_t)> gaussianHeaderBuffer,
	std::function<char* (size_t)> cacheBuffer,
	const int P, int D, int M,
	const float* background,
	const int width, int height,
	const float* means3D,
	const float* shs,
	const float* colors_precomp,
	const float* opacities,
	const float* scales,
	const float scale_modifier,
	const float* rotations,
	const float* cov3D_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const float* cam_pos,
	const float tan_fovx, float tan_fovy,
	const bool prefiltered,
	int* radii)
{

	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	size_t chunk_size = required<GeometryState>(P);
	char* chunkptr = geometryBuffer(chunk_size);
	GeometryState geomState = GeometryState::fromChunk(chunkptr, P);

	if (radii == nullptr)
	{
		radii = geomState.internal_radii;
	}

	dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	dim3 block(BLOCK_X, BLOCK_Y, 1);

	// Dynamically resize image-based auxiliary buffers during training
	int img_chunk_size = required<ImageState>(width * height);
	char* img_chunkptr = imageBuffer(img_chunk_size);
	ImageState imgState = ImageState::fromChunk(img_chunkptr, width * height);

	if (NUM_CHANNELS != 3 && colors_precomp == nullptr)
	{
		throw std::runtime_error("For non-RGB, provide precomputed Gaussian colors!");
	}

	// Run preprocessing per-Gaussian (transformation, bounding, conversion of SHs to RGB)
	FORWARD::preprocess(
		P, D, M,
		means3D,
		(glm::vec3*)scales,
		scale_modifier,
		(glm::vec4*)rotations,
		opacities,
		shs,
		geomState.clamped,
		cov3D_precomp,
		colors_precomp,
		viewmatrix, projmatrix,
		(glm::vec3*)cam_pos,
		width, height,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
		radii,
		geomState.means2D,
		geomState.depths,
		geomState.cov3D,
		geomState.rgb,
		geomState.conic_opacity,
		geomState.bbx_min,
		geomState.bbx_max,
		tile_grid,
		geomState.tiles_touched,
		prefiltered
	);

	// Compute prefix sum over full list of touched tile counts by Gaussians
	// E.g., [2, 3, 0, 2, 1] -> [2, 5, 5, 7, 8]
	hipcub::DeviceScan::InclusiveSum(geomState.scanning_space, geomState.scan_size,
		geomState.tiles_touched, geomState.point_offsets, P);

	// Retrieve total number of Gaussian instances to launch and resize aux buffers
	int num_rendered;
	CUDA_CHECK(hipMemcpy(&num_rendered, geomState.point_offsets + P - 1, sizeof(int), hipMemcpyDeviceToHost));

	int binning_chunk_size = required<BinningState>(num_rendered);
	char* binning_chunkptr = binningBuffer(binning_chunk_size);
	BinningState binningState = BinningState::fromChunk(binning_chunkptr, num_rendered);

	// For each instance to be rendered, produce adequate [ tile | depth ] key 
	// and corresponding dublicated Gaussian indices to be sorted
	duplicateWithKeys << <(P + 255) / 256, 256 >> > (
		P,
		geomState.means2D,
		geomState.depths,
		geomState.point_offsets,
		binningState.point_list_keys_unsorted,
		binningState.point_list_unsorted,
		radii,
		tile_grid
		);

	int bit = getHigherMsb(tile_grid.x * tile_grid.y);

	// Sort complete list of (duplicated) Gaussian indices by keys
	hipcub::DeviceRadixSort::SortPairs(
		binningState.list_sorting_space,
		binningState.sorting_size,
		binningState.point_list_keys_unsorted, binningState.point_list_keys,
		binningState.point_list_unsorted, binningState.point_list,
		num_rendered, 0, 32 + bit);

	CUDA_CHECK(hipMemset(imgState.ranges, 0, tile_grid.x * tile_grid.y * sizeof(uint2)));

	// Identify start and end of per-tile workloads in sorted list
	if (num_rendered > 0)
		identifyTileRanges << <(num_rendered + 255) / 256, 256 >> > (
			num_rendered,
			binningState.point_list_keys,
			imgState.ranges
			);

	const float* feature_ptr = colors_precomp != nullptr ? colors_precomp : geomState.rgb;

	// Create transmittance cache for flow rendering
	GaussianHeaderState gaussianHeaderState;
	CacheState cacheState;

	CudaRasterizer::allocateCache(
		P, tile_grid, block,
		imgState, binningState, geomState,
		gaussianHeaderBuffer, cacheBuffer,
		width, height,
		feature_ptr,
		background,
		geomState.depths,
		gaussianHeaderState,
		cacheState);

	// Let each tile blend its range of Gaussians independently in parallel
	FORWARD::renderCache(
		tile_grid, block,
		imgState.ranges,
		binningState.point_list,
		width, height, P,
		geomState.means2D,
		feature_ptr,
		geomState.conic_opacity,
		imgState.accum_alpha,
		imgState.n_contrib,
		background,
		geomState.depths,
		gaussianHeaderState.bbx_min,
		gaussianHeaderState.bbx_max,
		gaussianHeaderState.cache_offset,
		cacheState.t_value,
		cacheState.g_value);
}

void computeSumCUDA(uint32_t P, 
	const uint64_t* __restrict__ counts, 
	uint64_t* __restrict__ sum)
{
	void* d_temp_storage = nullptr;
	size_t temp_storage_bytes = 0;

	// Get the size of the temporary storage needed for the scan
	hipcub::DeviceReduce::Sum(
		d_temp_storage, temp_storage_bytes,
		counts, sum, P
	);

	if (temp_storage_bytes > 0)
	{
		CUDA_CHECK(hipMalloc(&d_temp_storage, temp_storage_bytes));
	}

	// Perform the reduction
	hipcub::DeviceReduce::Sum(
		d_temp_storage, temp_storage_bytes,
		counts, sum, P
	);

	// Free the temporary storage
	if (d_temp_storage != nullptr)
	{
		CUDA_CHECK(hipFree(d_temp_storage));
	}
}

void computePrefixSumCUDA(
	uint64_t* __restrict__ counts,
	uint64_t* __restrict__ offsets,
	uint32_t P)
{
	void* d_temp_storage = nullptr;
	size_t temp_storage_bytes = 0;

	// Get the size of the temporary storage needed for the scan
	hipcub::DeviceScan::ExclusiveSum(
		d_temp_storage, temp_storage_bytes,
		counts, offsets, P
	);

	if (temp_storage_bytes > 0)
	{
		CUDA_CHECK(hipMalloc(&d_temp_storage, temp_storage_bytes));
	}

	// Perform the prefix sum
	hipcub::DeviceScan::ExclusiveSum(
		d_temp_storage, temp_storage_bytes,
		counts, offsets, P
	);

	// Free the temporary storage
	if (d_temp_storage != nullptr)
	{
		CUDA_CHECK(hipFree(d_temp_storage));
	}
}

void CudaRasterizer::allocateCache(
	int P, dim3 tile_grid, dim3 block,
	const ImageState& imgState,
	const BinningState& binningState,
	const GeometryState& geomState,
	std::function<char* (size_t)> gaussianHeaderBuffer,
	std::function<char* (size_t)> cacheBuffer,
	const int width, const int height,
	const float* feature_ptr,
	const float* bg_color,
	const float* depth,
	GaussianHeaderState& out_gaussianHeaderState,
	CacheState& out_cacheState	
)
{
	// Allocate temporary variables for computing the buffer sizes dynamically
	uint64_t* cache_counts_per_gaussian; // Number of T, G values per Gaussian
	uint64_t* cache_offsets; // Offsets computed from cache counts

	CUDA_CHECK(hipMalloc((void**)&cache_counts_per_gaussian, P * sizeof(uint64_t)));
	CUDA_CHECK(hipMalloc((void**)&cache_offsets, P * sizeof(uint64_t)));

	printf("Test1\n");

	FORWARD::computeCacheLayout(
		P, width, height,
		geomState.bbx_min,
		geomState.bbx_max,
		cache_counts_per_gaussian
	);

	printf("Test2\n");

	// Compute the total number of cache entries needed
	uint64_t total_cache_count = 0;
	uint64_t* d_total_cache_count;
	CUDA_CHECK(hipMalloc((void**)&d_total_cache_count, sizeof(uint64_t)));
	computeSumCUDA(P, cache_counts_per_gaussian, d_total_cache_count);
	CUDA_CHECK(hipMemcpy(&total_cache_count, d_total_cache_count, sizeof(uint64_t), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipFree(d_total_cache_count));

	printf("Test3\n");

	// Compute prefix sum over gaussians
	computePrefixSumCUDA(cache_counts_per_gaussian, cache_offsets, P);

	printf("Test4\n");

	// Allocate data structure
	size_t gaussian_header_chunk_size = required<GaussianHeaderState>(P);
	char* gaussian_header_chunkptr = gaussianHeaderBuffer(gaussian_header_chunk_size);
	out_gaussianHeaderState = GaussianHeaderState::fromChunk(gaussian_header_chunkptr, P);

	size_t cache_chunk_size = required<CacheState>(total_cache_count);
	char* cache_chunkptr = cacheBuffer(cache_chunk_size);
	out_cacheState = CacheState::fromChunk(cache_chunkptr, total_cache_count);

	printf("Test5\n");

	hipStream_t stream;
	CUDA_CHECK(hipStreamCreate(&stream));

	CUDA_CHECK(hipMemcpyAsync(
		out_gaussianHeaderState.bbx_min, geomState.bbx_min,
		P * sizeof(uint2), hipMemcpyDeviceToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(
		out_gaussianHeaderState.bbx_max, geomState.bbx_max,
		P * sizeof(uint2), hipMemcpyDeviceToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(
		out_gaussianHeaderState.cache_offset, cache_offsets,
		P * sizeof(uint64_t), hipMemcpyDeviceToDevice, stream));
	CUDA_CHECK(hipMemsetAsync(out_cacheState.t_value, 0, total_cache_count * sizeof(float), stream));
	CUDA_CHECK(hipMemsetAsync(out_cacheState.g_value, 0, total_cache_count * sizeof(float), stream));

	CUDA_CHECK(hipStreamSynchronize(stream));
	CUDA_CHECK(hipStreamDestroy(stream));

	// Free temporary variables
	CUDA_CHECK(hipFree(cache_counts_per_gaussian));
	CUDA_CHECK(hipFree(cache_offsets));

	CUDA_CHECK(hipDeviceSynchronize());

	uint2* h_bbx_min;
	uint2* h_bbx_max;
	uint64_t *h_cache_offsets;
	CUDA_CHECK(hipHostMalloc((void**)&h_cache_offsets, P * sizeof(uint64_t)));
	CUDA_CHECK(hipHostMalloc((void**)&h_bbx_min, P * sizeof(uint2)));
	CUDA_CHECK(hipHostMalloc((void**)&h_bbx_max, P * sizeof(uint2)));
	CUDA_CHECK(hipMemcpy(h_cache_offsets, out_gaussianHeaderState.cache_offset, P * sizeof(uint64_t), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(h_bbx_min, out_gaussianHeaderState.bbx_min, P * sizeof(uint2), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(h_bbx_max, out_gaussianHeaderState.bbx_max, P * sizeof(uint2), hipMemcpyDeviceToHost));

	printf("---------------------------------------------\n");
	printf("Cache layout for %d Gaussians:\n", P);
	printf("Total cache entries: %u\n", total_cache_count);
	printf("Cache size: %zu bytes\n", cache_chunk_size);
	printf("---------------------------------------------\n");

	// Iterate over all Gaussian headers
	for (int i = 0; i < 1000; i++)
	{
		// Compute the bounding box for the Gaussian
		uint2 bbx_min = h_bbx_min[i];
		uint2 bbx_max = h_bbx_max[i];	

		// Compute the cache offset for this Gaussian
		uint32_t cache_offset = h_cache_offsets[i];

		// Print the bounding box and cache offset
		printf("Gaussian %d: BBX Min: (%u, %u), BBX Max: (%u, %u), Cache Offset: %u\n",
			i, bbx_min.x, bbx_min.y,
			bbx_max.x, bbx_max.y, cache_offset);
	}

	CUDA_CHECK(hipHostFree(h_bbx_min));
	CUDA_CHECK(hipHostFree(h_bbx_max));
	CUDA_CHECK(hipHostFree(h_cache_offsets));
}

// Forward rendering procedure for differentiable rasterization
// of Gaussians.
int CudaRasterizer::FlowRasterizer::forward(
	std::function<char* (size_t)> geometryBuffer,
	std::function<char* (size_t)> flowBuffer,
	std::function<char* (size_t)> binningBuffer,
	std::function<char* (size_t)> imageBuffer,
	const int P, int D, int M,
	const float* background,
	const int width, int height,
	const float* means3D,
	const float* prev_means3D,
	const float* shs,
	const float* colors_precomp,
	const float* opacities,
	const float* prev_opacities,
	const float* scales,
	const float* prev_scales,
	const float scale_modifier,
	const float* rotations,
	const float* prev_rotations,
	const float* cov3D_precomp,
	const float* prev_cov3D_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const float* cam_pos,
	char* gaussianHeaderBuffer,
	const uint32_t gaussianHeaderBufferSize,
	char* cacheBuffer,
	const uint32_t cacheBufferSize,
	const float tan_fovx, float tan_fovy,
	const bool prefiltered,
	float* out_color,
	float* out_depth,
	float* out_flow,
	int* radii)
{
	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	size_t chunk_size = required<GeometryState>(P);
	char* chunkptr = geometryBuffer(chunk_size);
	GeometryState geomState = GeometryState::fromChunk(chunkptr, P);

	size_t flow_chunk_size = required<FlowState>(P);
	char* flow_chunkptr = flowBuffer(flow_chunk_size);
	FlowState flowState = FlowState::fromChunk(flow_chunkptr, P);

	if (radii == nullptr)
	{
		radii = geomState.internal_radii;
	}

	dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	dim3 block(BLOCK_X, BLOCK_Y, 1);

	// Dynamically resize image-based auxiliary buffers during training
	int img_chunk_size = required<ImageState>(width * height);
	char* img_chunkptr = imageBuffer(img_chunk_size);
	ImageState imgState = ImageState::fromChunk(img_chunkptr, width * height);

	if (NUM_CHANNELS != 3 && colors_precomp == nullptr)
	{
		throw std::runtime_error("For non-RGB, provide precomputed Gaussian colors!");
	}

	// Run preprocessing per-Gaussian (transformation, bounding, conversion of SHs to RGB)
	FORWARD::FLOW::preprocess(
		P, D, M,
		means3D,
		prev_means3D,
		(glm::vec3*)scales,
		(glm::vec3*)prev_scales,
		scale_modifier,
		(glm::vec4*)rotations,
		(glm::vec4*)prev_rotations,
		opacities,
		prev_opacities,
		shs,
		geomState.clamped,
		cov3D_precomp,
		prev_cov3D_precomp,
		colors_precomp,
		viewmatrix, projmatrix,
		(glm::vec3*)cam_pos,
		width, height,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
		radii,
		geomState.means2D,
		flowState.prev_means2D,
		geomState.depths,
		geomState.cov3D,
		geomState.rgb,
		geomState.conic_opacity,
		tile_grid,
		geomState.tiles_touched,
		flowState.prev_cov2D_opacity,
		flowState.sqrt_conic,
		flowState.prev_sqrt_cov2D,
		prefiltered
	);

	// Compute prefix sum over full list of touched tile counts by Gaussians
	// E.g., [2, 3, 0, 2, 1] -> [2, 5, 5, 7, 8]
	hipcub::DeviceScan::InclusiveSum(geomState.scanning_space, geomState.scan_size,
		geomState.tiles_touched, geomState.point_offsets, P);

	// Retrieve total number of Gaussian instances to launch and resize aux buffers
	int num_rendered;
	CUDA_CHECK(hipMemcpy(&num_rendered, geomState.point_offsets + P - 1, sizeof(int), hipMemcpyDeviceToHost));

	int binning_chunk_size = required<BinningState>(num_rendered);
	char* binning_chunkptr = binningBuffer(binning_chunk_size);
	BinningState binningState = BinningState::fromChunk(binning_chunkptr, num_rendered);

	// For each instance to be rendered, produce adequate [ tile | depth ] key 
	// and corresponding dublicated Gaussian indices to be sorted
	duplicateWithKeys << <(P + 255) / 256, 256 >> > (
		P,
		geomState.means2D,
		geomState.depths,
		geomState.point_offsets,
		binningState.point_list_keys_unsorted,
		binningState.point_list_unsorted,
		radii,
		tile_grid
		);

	int bit = getHigherMsb(tile_grid.x * tile_grid.y);

	// Sort complete list of (duplicated) Gaussian indices by keys
	hipcub::DeviceRadixSort::SortPairs(
		binningState.list_sorting_space,
		binningState.sorting_size,
		binningState.point_list_keys_unsorted, binningState.point_list_keys,
		binningState.point_list_unsorted, binningState.point_list,
		num_rendered, 0, 32 + bit);

	CUDA_CHECK(hipMemset(imgState.ranges, 0, tile_grid.x * tile_grid.y * sizeof(uint2)));

	// Identify start and end of per-tile workloads in sorted list
	if (num_rendered > 0)
		identifyTileRanges << <(num_rendered + 255) / 256, 256 >> > (
			num_rendered,
			binningState.point_list_keys,
			imgState.ranges
			);


	// Cast the cache buffers to the appropriate types
	GaussianHeaderState gaussianHeaderState = GaussianHeaderState::fromChunk(
		gaussianHeaderBuffer, gaussianHeaderBufferSize);
	CacheState cacheState = CacheState::fromChunk(
		cacheBuffer, cacheBufferSize);

	CUDA_CHECK(hipDeviceSynchronize());

	// Let each tile blend its range of Gaussians independently in parallel
	const float* feature_ptr = colors_precomp != nullptr ? colors_precomp : geomState.rgb;
	FORWARD::FLOW::render(
		tile_grid, block,
		imgState.ranges,
		binningState.point_list,
		P, width, height,
		geomState.means2D,
		flowState.prev_means2D,
		feature_ptr,
		geomState.conic_opacity,
		flowState.prev_cov2D_opacity,
		flowState.sqrt_conic,
		flowState.prev_sqrt_cov2D,
		imgState.accum_alpha,
		imgState.n_contrib,
		background,
		out_color,
		geomState.depths,
		out_depth,
		gaussianHeaderState.bbx_min,
		gaussianHeaderState.bbx_max,
		gaussianHeaderState.cache_offset,
		cacheState.t_value,
		cacheState.g_value,
		out_flow);

	CUDA_CHECK(hipDeviceSynchronize());

	return num_rendered;
}